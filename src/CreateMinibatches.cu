#include "hip/hip_runtime.h"
#include "Matrix.hpp"
#include <iostream>
#include <fstream>
#include <string>

__device__ __host__ int littleToBigEndian(int num) {
    #ifdef LITTLE_ENDIAN
        return (num & 0xFF) << 24 | (num & 0xFF00) << 8 | ((num >> 8) & 0xFF00) | ((num >> 24) & 0xFF);
    #else
        return num;
    #endif
}

void saveMinibatch(Matrix_F& minibatchData, Matrix_F& minibatchLabels, std::string& outputPath, int minibatchNum) {
    std::string minibatchSaveFile = outputPath + "/" + std::to_string(minibatchNum) + ".minibatch";
    std::cout << "Saving minibatch " << minibatchNum << " to " << minibatchSaveFile << '\n';
    // Open a file for saving.
    std::ofstream outputFile(minibatchSaveFile, std::ios::binary);
    // Save!
    minibatchData.save(outputFile);
    minibatchLabels.save(outputFile);
    outputFile.close();

}

void parseHeader(std::ifstream& dataFile, std::ifstream& labelFile, int& magicNumberData,
    int& magicNumberLabels, int& numItems, int& rows, int& cols, int numMinibatches, int& minibatchSize) {
    // Read magic numbers.
    dataFile.read(reinterpret_cast<char*>(&magicNumberData), sizeof magicNumberData);
    labelFile.read(reinterpret_cast<char*>(&magicNumberLabels), sizeof magicNumberLabels);
    // Figure out how much data is in the file
    dataFile.read(reinterpret_cast<char*>(&numItems), sizeof numItems);
    labelFile.read(reinterpret_cast<char*>(&numItems), sizeof numItems);
    // Get dimensions of images.
    dataFile.read(reinterpret_cast<char*>(&rows), sizeof rows);
    dataFile.read(reinterpret_cast<char*>(&cols), sizeof cols);
    // Convert everything to big endian.
    numItems = littleToBigEndian(numItems);
    rows = littleToBigEndian(rows);
    cols = littleToBigEndian(cols);
    // Figure out the size of a minibatch.
    minibatchSize = std::ceil(numItems / (float) numMinibatches);
    // Output
    std::cout << "Data magic number: " << littleToBigEndian(magicNumberData) << '\n';
    std::cout << "Label magic number: " << littleToBigEndian(magicNumberLabels) << '\n';
    std::cout << "Number of Items: " << numItems << '\n';
    std::cout << "Dimensions: " << rows << "x" << cols << '\n';
    std::cout << "Minibatch Size: " << minibatchSize << '\n';
}

void parseData(std::ifstream& dataFile, std::ifstream& labelFile, Matrix_UC& minibatchDataRaw, Matrix_UC& minibatchLabelsRaw) {
    dataFile.read(reinterpret_cast<char*>(&minibatchDataRaw[0]), sizeof(minibatchDataRaw[0]) * minibatchDataRaw.size());
    labelFile.read(reinterpret_cast<char*>(&minibatchLabelsRaw[0]), sizeof(minibatchLabelsRaw[0]) * minibatchLabelsRaw.size());
}

void processData(Matrix_F& minibatchData, Matrix_UC& minibatchDataRaw) {
    minibatchData = (255 - minibatchDataRaw.asType<float>()) / 255;
}

void processLabels(Matrix_F& minibatchLabels, Matrix_UC& minibatchLabelsRaw) {
    for (int row = 0; row < minibatchLabelsRaw.numRows(); ++row) {
        minibatchLabels.at(row, minibatchLabelsRaw[row]) = 1.0;
    }
}

// Translates the MNIST dataset into a matrix friendly format.
int main(int argc, char const *argv[]) {
    std::string dataPath, labelPath, outputPath;
    int numMinibatches = 1;
    try {
        // Get the raw data file and then the labels.
        dataPath = argv[1];
        labelPath = argv[2];
        outputPath = argv[3];
        numMinibatches = (argc % 2) ? std::stoi(argv[argc - 1]) : numMinibatches;
    } catch (const std::exception& e) {
        std::cout << "Usage: " << argv[0] << " DATA-FILE LABELS-FILE OUTPUT-DIR [# MINIBATCHES]" << '\n';
        return 1;
    }
    // Metadata
    int magicNumberData, magicNumberLabels, numItems, rows, cols, minibatchSize;
    // Open files
    std::ifstream dataFile(dataPath, std::ios::binary);
    std::ifstream labelFile(labelPath, std::ios::binary);
    // Load into matrices.
    if (dataFile.is_open() && labelFile.is_open()) {
        parseHeader(dataFile, labelFile, magicNumberData, magicNumberLabels, numItems, rows, cols, numMinibatches, minibatchSize);
        // Data contains images, labels are 10 values with 1 of them equal to 1.0.
        Matrix_UC minibatchDataRaw(minibatchSize, rows * cols), minibatchLabelsRaw(minibatchSize, 1);
        Matrix_F minibatchData(minibatchSize, rows * cols), minibatchLabels(minibatchSize, 10);
        // Loop over all minibatches.
        for (int i = 0; i < numMinibatches - 1; ++i) {
            // Read in data 1 minibatch at a time.
            parseData(dataFile, labelFile, minibatchDataRaw, minibatchLabelsRaw);
            // Process the minibatch.
            processData(minibatchData, minibatchDataRaw);
            processLabels(minibatchLabels, minibatchLabelsRaw);
            // Save.
            saveMinibatch(minibatchData, minibatchLabels, outputPath, i);
        }

        // Handle leftover items.
        int itemsRemaining = numItems - minibatchSize * (numMinibatches - 1);
        minibatchDataRaw = Matrix_UC(itemsRemaining, rows * cols);
        minibatchLabelsRaw = Matrix_UC(itemsRemaining, 1);
        minibatchData = Matrix_F(itemsRemaining, rows * cols);
        minibatchLabels = Matrix_F(itemsRemaining, 10);
        // Read data and labels for last minibatch.
        parseData(dataFile, labelFile, minibatchDataRaw, minibatchLabelsRaw);
        // Process it.
        processData(minibatchData, minibatchDataRaw);
        processLabels(minibatchLabels, minibatchLabelsRaw);
        // Save.
        saveMinibatch(minibatchData, minibatchLabels, outputPath, numMinibatches - 1);
    } else {
        throw std::invalid_argument("Could not open data.");
    }
}
